#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <time.h>
#include <random>
#include <cmath>
#include <chrono>
#include <thread>
#include <algorithm>

constexpr unsigned long long kNumberOfBits = 10;
constexpr unsigned long long kNumberOfSequences = 10000;
constexpr unsigned long long kNumberOfPairs = (kNumberOfSequences * (kNumberOfSequences - 1)) / 2;

class Sequence {
	char bytes[(kNumberOfBits / 64 + (!!(kNumberOfBits % 64))) * 8];
public:
	__host__ __device__ char* getBytes() {
		return bytes;
	}
};

class Result {
	char bytes[(kNumberOfPairs / 64 + (!!(kNumberOfPairs % 64))) * 8];
public:
	__host__ __device__ char* getBytes() {
		return bytes;
	}
};

void generateInput(Sequence* bits);
void printSequence(Sequence& sequence);
__host__ __device__ unsigned long long* getWord(char* bits, unsigned long long j);
__host__ __device__ unsigned long long* getWord32(char* bits, unsigned long long j);
__host__ __device__ char checkDistance(Sequence& sequence1, Sequence& sequence2, unsigned long long nrOfBits);
__host__ __device__ inline void k2ij(unsigned long long  k, unsigned long long* i, unsigned long long* j);
__host__ __device__ unsigned long long ij2k(unsigned long long i, unsigned long long j);
__host__ __device__ inline void SetBit(char* array, unsigned long long index, char value)
{
	array[index / 8] = (array[index / 8] & (~(1 << (index % 8)))) | ((!!value) << (index % 8));
}
__host__ __device__ inline char GetBit(char* array, unsigned long long index)
{
	return array[index / 8] >> (index % 8) & 1;
}
__global__ void hammingGPU(Sequence* d_sequences, char* results, unsigned long long nrOfSeq, unsigned long long nrOfBits,
	unsigned long long offset = 0);
__global__ void hammingGPUPairs(Sequence* d_sequences, char* results, unsigned long long nrOfSeq, unsigned long long nrOfBits,
	unsigned long long offset = 0);

//naprawi� tooo:
__global__ void hammingGPU(Sequence* d_sequences, char* results, unsigned long long nrOfSeq, unsigned long long nrOfBits,
	unsigned long long offset) {
	unsigned long long threadId = threadIdx.x + blockIdx.x * blockDim.x + offset;
	for (unsigned long long i = 0; i < threadId; i++) {
		//SetBit(results->getBytes(), ij2k(i, threadId), checkDistance(d_sequences[threadId], d_sequences[i], nrOfBits));
		results[ij2k(threadId, i)] = checkDistance(d_sequences[threadId], d_sequences[i], nrOfBits);
	}
}

__global__ void hammingGPUPairs(Sequence* d_sequences, char* results, unsigned long long nrOfSeq, unsigned long long nrOfBits,
	unsigned long long offset) {
	unsigned long long threadId = threadIdx.x + blockIdx.x * blockDim.x + offset;
		//SetBit(results->getBytes(), ij2k(i, threadId), checkDistance(d_sequences[threadId], d_sequences[i], nrOfBits));
	unsigned long long s1, s2;
	k2ij(threadId, &s1, &s2);
	results[threadId] = checkDistance(d_sequences[s1], d_sequences[s2], nrOfBits);
}

std::vector<std::pair<unsigned long long, unsigned long long>> getPairs(char* results, unsigned long long* sum) {
	*sum = 0;
	std::vector<std::pair<unsigned long long, unsigned long long>> pairs;
	for (int i = 0; i < kNumberOfPairs; i++) {
		if (results[i] == 1) {
			unsigned long long s1, s2;
			k2ij(i, &s1, &s2);
			if (s1 != s2) {
				*sum += 1;
				pairs.push_back(std::make_pair(s1, s2));
			}
		}
	}
	return pairs;
}

auto pairsGPU(Sequence* h_sequence) {
	Sequence* d_sequence;
	char *h_results, *d_results;
	h_results = new char[kNumberOfPairs];
	hipMalloc(&d_sequence, sizeof(Sequence) * kNumberOfSequences);
	hipMemcpy(d_sequence, h_sequence, sizeof(Sequence) * kNumberOfSequences, hipMemcpyHostToDevice);
	hipMalloc(&d_results, sizeof(char) * kNumberOfPairs);
	hipMemcpy(d_results, h_results, sizeof(char) * kNumberOfPairs, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	hipEventSynchronize(start);
	//hammingGPUPairs << < kNumberOfPairs / 1024, 1024 >> > (d_sequence, d_results, kNumberOfSequences, kNumberOfBits);
	//if (kNumberOfPairs % 1024) {
	//	hammingGPUPairs << < 1, kNumberOfPairs % 1024 >> > (d_sequence, d_results, kNumberOfSequences, kNumberOfBits,
	//		kNumberOfPairs - kNumberOfPairs % 1024);
	//}
	hammingGPU << < kNumberOfSequences / 1024, 1024 >> > (d_sequence, d_results, kNumberOfSequences, kNumberOfBits);
	if (kNumberOfSequences % 1024) {
		hammingGPU << < 1, kNumberOfSequences % 1024 >> > (d_sequence, d_results, kNumberOfSequences, kNumberOfBits,
			kNumberOfSequences - kNumberOfSequences % 1024);
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);

	hipMemcpy(h_results, d_results, sizeof(char) * kNumberOfPairs, hipMemcpyDeviceToHost);

	unsigned long long sum = 0;
	const auto& pairs = getPairs(h_results, &sum);
	for (const auto& pair : pairs) {
		//std::cout << "Sequence 1: " << pair.first <<
		//	", sequence 2: " << pair.second << std::endl;
		std::cout << "Sequence 1: ";
		printSequence(h_sequence[pair.first]);
		std::cout << " , sequence 2: ";
		printSequence(h_sequence[pair.second]);
		int ret = checkDistance(h_sequence[pair.first], h_sequence[pair.second], kNumberOfBits);
		std::cout << std::endl;
	}
	std::cout << "sum gpu: " << sum << ", time: " << time << std::endl;
	return pairs;
}

auto pairsCPU(Sequence* sequences) {
	Sequence* d_sequence;
	char *results;
	results = new char[kNumberOfPairs];
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	hipEventSynchronize(start);
	for (int i = 0; i < kNumberOfSequences; i++) {
		for (int j = 0; j < i; j++) {
			//SetBit(results->getBytes(), ij2k(j, i), checkDistance(sequences[i], sequences[j], kNumberOfBits));
			results[ij2k(i, j)] = checkDistance(sequences[i], sequences[j], kNumberOfBits);
		}
	}
	//for (unsigned long long i = 0; i < kNumberOfPairs; i++) {
	//	unsigned long long s1, s2;
	//	k2ij(i, &s1, &s2);
	//	results[i] = checkDistance(sequences[s1], sequences[s2], kNumberOfBits);
	//}
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float time;
	hipEventElapsedTime(&time, start, stop);

	unsigned long long sum = 0;
	//std::vector<std::pair<unsigned long long, unsigned long long>> pairs;
	//for (int i = 0; i < kNumberOfPairs; i++) {
	//	if ((short int)GetBit(results->getBytes(), i) == 1) {
	//		sum += 1;
	//		unsigned long long s1, s2;
	//		k2ij(i, &s1, &s2);
	//		pairs.push_back(std::make_pair(s1, s2));
	//	}
	//}
	const auto& pairs = getPairs(results, &sum);
	//for (const auto& pair : hmm) {
	//	//std::cout << "Sequence 1: " << pair.first <<
	//	//	", sequence 2: " << pair.second<< std::endl;
	//	std::cout << "Sequence 1: ";
	//	printSequence(sequences[pair.first]);
	//	std::cout << " , sequence 2: ";
	//	printSequence(sequences[pair.second]);
	//	int ret = checkDistance(sequences[pair.first], sequences[pair.second], kNumberOfBits);
	//	std::cout << std::endl;
	//}

	for (const auto& pair : pairs) {
		//std::cout << "Sequence 1: " << pair.first <<
		//	", sequence 2: " << pair.second<< std::endl;
		//std::cout << "Sequence 1: ";
		//printSequence(sequences[pair.first]);
		//std::cout << " , sequence 2: ";
		//printSequence(sequences[pair.second]);
		//int ret = checkDistance(sequences[pair.first], sequences[pair.second], kNumberOfBits);
		//std::cout << std::endl;
	}
	std::cout << "sum cpu: " << sum << ", time: " << time << std::endl;
	return pairs;
}

int main() {
	Sequence* sequences = new Sequence[kNumberOfSequences];
	generateInput(sequences);
	auto pairs1 = pairsCPU(sequences);
	auto pairs2 = pairsGPU(sequences);
	for (const auto& pair : pairs1) {
		if (find(pairs2.begin(), pairs2.end(), pair) == pairs2.end()) {
			std::cout << "pair not found in gpu: " << "s1: " << pair.first << " ";
			printSequence(sequences[pair.first]);
			std::cout << ", " << "s2: " << pair.second << " ";
			printSequence(sequences[pair.second]);
			std::cout << std::endl;
		}
	}
    return 0;
}

void generateInput(Sequence* bits) {
	std::mt19937_64 random;
	int seed = std::random_device()();
	random.seed(seed);
	for (int i = 0; i < kNumberOfSequences; i++) {
		for (int j = 0; j < kNumberOfBits / 64; j++) {
			*getWord(bits[i].getBytes(), j) = random();
		}
		*getWord(bits[i].getBytes(), kNumberOfBits / 64) = random() >> (64 - (kNumberOfBits % 64));
	}
}

void printSequence(Sequence& sequence) {
	for (unsigned long long i = 0; i < kNumberOfBits; i++) {
		std::cout << (sequence.getBytes()[i / 8] >> (i % 8) & 1);
	}
}

__host__ __device__ unsigned long long* getWord(char* bits, unsigned long long j) {
	return (unsigned long long*)(bits + j * 64 / 8);
}

__host__ __device__ unsigned long long* getWord32(char* bits, unsigned long long j) {
	return (unsigned long long*)(bits + j * 32 / 8);
}

__host__ __device__ char checkDistance(Sequence& sequence1, Sequence& sequence2, unsigned long long nrOfBits) {
	int diff = 0;
	for (int j = 0; j < (nrOfBits + 63) / 64; ++j) {
		unsigned long long int a, b, xor;
		a = *(getWord(sequence1.getBytes(), j));
		b = *(getWord(sequence2.getBytes(), j));
		xor = a ^ b;
		diff += xor == 0 ? 0 : (xor & (xor -1) ? 2 : 1);
		if (diff > 1) {
			return 0;
		}
	}
	return !!diff;
}

__host__ __device__ inline void k2ij(unsigned long long  k, unsigned long long* i, unsigned long long* j) {
	*i = (unsigned int)ceilf((0.5f * (-1 + sqrtf(1 + 8 * (k + 1)))));
	*j = (unsigned int)((k + 1) - 0.5 * (*i) * ((*i) - 1)) - 1;
}

__host__ __device__ unsigned long long ij2k(unsigned long long i, unsigned long long j) {
	return i * (i - 1) / 2 + j;
}
